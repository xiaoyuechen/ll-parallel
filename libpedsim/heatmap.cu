#include "hip/hip_runtime.h"
#include <memory>
#include <hip/hip_runtime.h>
#include "ped_model.h"
#include <stdio.h>


#define BLOCK_NUMBER 4
#define BLOCK_WIDTH 256

namespace Ped {
  __global__ void InitSHeatmap(int* shm, int** scaled_heatmap) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    scaled_heatmap[tid] = shm + SCALED_SIZE * tid;
  }

  __global__ void InitHeatmap(int* hm, int** heatmap) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    heatmap[tid] = hm + SIZE * tid;
  }

  __global__ void CudaPrint(int* hm) {
    printf("%d\n",hm[255]);
  }

  void Model::setupHeatmapCuda() {
    // hipStream_t s[6];
    // for(int i = 0; i != 6; ++i) {
    //     hipStreamCreate(s + i);
    // }

    int *hm, *shm, *bhm;

    hipMalloc(&hm, SIZE * SIZE * sizeof(int));
    hipMalloc(&shm, SCALED_SIZE * SCALED_SIZE * sizeof(int));
    hipMalloc(&heatmap, SIZE * sizeof(int*));
    hipMalloc(&scaled_heatmap, SCALED_SIZE * sizeof(int*));

    //we need to calculate them on GPU as well?
    hipMalloc(&desired_xs, agents.size() * sizeof(int));
    hipMalloc(&desired_ys, agents.size() * sizeof(int));

    hipHostMalloc(&blurred_heatmap, SCALED_SIZE * sizeof(int*));
    hipHostMalloc(&bhm, SCALED_SIZE * SCALED_SIZE * sizeof(int));
    hipMemset(hm, 0, SIZE * SIZE);
    hipMemset(shm, 0, SCALED_SIZE * SCALED_SIZE);
    hipMemset(bhm, 1, SCALED_SIZE * SCALED_SIZE);


    InitHeatmap<<<1,SIZE>>>(hm, heatmap);
    hipDeviceSynchronize();

    InitSHeatmap<<<CELLSIZE,SIZE>>>(shm, scaled_heatmap);
    hipDeviceSynchronize();

    for (int i = 0; i < SCALED_SIZE; i++) {
      blurred_heatmap[i] = bhm + SCALED_SIZE * i;
    }
  }


  __global__ void heatFades(int** heatmap) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (int row = 0; row < SIZE; row++) {
      heatmap[row][tid] = (int)round(heatmap[row][tid] * 0.80);
    }
  }

  __global__ void coloringTheMap(int** heatmap, const int agents, int* desired_xs, int* desired_ys) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid > agents) return;
    if(desired_xs[tid]<0 || desired_xs[tid]>SIZE || 
        desired_ys[tid]<0 || desired_ys[tid]>SIZE) 
        return;

    int i = desired_ys[tid];
    atomicAdd(&heatmap[desired_xs[tid]][i], 40);
  }

  __global__ void coloringTheMap1(int** heatmap) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for(int i = 0; i < SIZE; ++i)
      atomicMin(&heatmap[tid][i], 255);
  }

  __global__ void scalingTheMap(int** heatmap, int** scaled_heatmap){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (int x = 0; x < SIZE; x++) {
      int value = heatmap[tid][x];
      for (int cellY = 0; cellY < CELLSIZE; cellY++) {
        for (int cellX = 0; cellX < CELLSIZE; cellX++) {
          scaled_heatmap[tid * CELLSIZE + cellY][x * CELLSIZE + cellX] = value;
        }
      }
    }
  }

  __global__ void filteringTheMap(int** scaled_heatmap, int** blurred_heatmap, const int w[5][5]){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Apply gaussian blurfilter
    if((tid > 1) && (tid < SCALED_SIZE - 2))
      for (int j = 2; j < SCALED_SIZE - 2; j++) {
        int sum = 0;
        for (int k = -2; k < 3; k++) {
          for (int l = -2; l < 3; l++) {
            sum += w[2 + k][2 + l] * scaled_heatmap[tid + k][j + l];
          }
        }
        int value = sum / 273;
        // printf("%d\n", value);
        auto temp = 0x00FF0000 | value << 24;
        // blurred_heatmap[tid][j] = temp;
      }

  }

  void Model::updateHeatmapCuda() {
    float time1, time2, time3, time4;
    hipEvent_t fade_start, fade_stop;
    hipEventCreate(&fade_start);
    hipEventCreate(&fade_stop);
    hipEventRecord(fade_start, 0);

    heatFades<<<1, SIZE>>>(heatmap);

    hipEventRecord(fade_stop, 0);
    hipEventSynchronize(fade_stop);
    hipEventElapsedTime(&time1, fade_start, fade_stop);
    hipEventDestroy(fade_start);
    hipEventDestroy(fade_stop);

    // ////////////////////////////////////////////////////
    hipMemcpyAsync(desired_xs, (*agent_soa).xs, agents.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpyAsync(desired_ys, (*agent_soa).ys, agents.size() * sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t coloring_start, coloring_stop;
    hipEventCreate(&coloring_start);
    hipEventCreate(&coloring_stop);
    hipEventRecord(coloring_start, 0);

    coloringTheMap<<<CELLSIZE, SIZE>>>(heatmap, agents.size(), desired_xs, desired_ys);

    hipEventRecord(coloring_stop, 0);
    hipEventSynchronize(coloring_stop);
    hipEventElapsedTime(&time2, coloring_start, coloring_stop);
    hipEventDestroy(coloring_start);
    hipEventDestroy(coloring_stop);

    coloringTheMap1<<<1, SIZE>>>(heatmap);
    hipDeviceSynchronize();

    ///////////////////////////////////////////////////

    hipEvent_t scaling_start, scaling_stop;
    hipEventCreate(&scaling_start);
    hipEventCreate(&scaling_stop);
    hipEventRecord(scaling_start, 0);

    scalingTheMap<<<1,SIZE>>>(heatmap, scaled_heatmap);

    hipEventRecord(scaling_stop, 0);
    hipEventSynchronize(scaling_stop);
    hipEventElapsedTime(&time3, scaling_start, scaling_stop);
    hipEventDestroy(scaling_start);
    hipEventDestroy(scaling_stop);

    ///////////////////////////////////////////////////

    const int w[5][5] = {{1, 4, 7, 4, 1},
    {4, 16, 26, 16, 4},
    {7, 26, 41, 26, 7},
    {4, 16, 26, 16, 4},
    {1, 4, 7, 4, 1}};

    hipEvent_t filtering_start, filtering_stop;
    hipEventCreate(&filtering_start);
    hipEventCreate(&filtering_stop);
    hipEventRecord(filtering_start, 0);

    filteringTheMap<<<1,SIZE>>>(scaled_heatmap, blurred_heatmap, w);

    hipEventRecord(filtering_stop, 0);
    hipEventSynchronize(filtering_stop);
    hipEventElapsedTime(&time4, filtering_start, filtering_stop);
    hipEventDestroy(filtering_start);
    hipEventDestroy(filtering_stop);
  }

}  // namespace Ped
