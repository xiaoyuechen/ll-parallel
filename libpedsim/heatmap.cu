#include "hip/hip_runtime.h"
#include <memory>
#include <hip/hip_runtime.h>
#include "ped_model.h"
#include <stdio.h>

#define BLOCK_NUMBER 4
#define BLOCK_WIDTH 256

namespace Ped {
  __global__ void InitSBHeatmap(int* bhm, int* shm, int** scaled_heatmap) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    scaled_heatmap[tid] = shm + SCALED_SIZE * tid;
  }

  void Model::setupHeatmapCuda() {
    // hipStream_t s[6];
    // for(int i = 0; i != 6; ++i) {
    //     hipStreamCreate(s + i);
    // }

    int *hm, *shm, *bhm;
    
    hipMalloc(&hm, SIZE * SIZE * sizeof(int));
    hipMalloc(&shm, SCALED_SIZE * SCALED_SIZE * sizeof(int));
    hipMalloc(&bhm, SCALED_SIZE * SCALED_SIZE * sizeof(int));
    hipMalloc(&heatmap, SIZE * sizeof(int*));
    hipMalloc(&scaled_heatmap, SCALED_SIZE * sizeof(int*));

    //we need to calculate them on GPU as well?
    hipMalloc(&desired_xs, 256 * sizeof(int));
    hipMalloc(&desired_ys, 256 * sizeof(int));

    hipHostMalloc(&blurred_heatmap, SCALED_SIZE * sizeof(int*));
    hipMemset(hm, 0, SIZE * SIZE);
    hipMemset(shm, 0, SCALED_SIZE * SCALED_SIZE);
    hipMemset(bhm, 0, SCALED_SIZE * SCALED_SIZE);

    InitSBHeatmap<<<CELLSIZE,SIZE>>>(bhm, shm, scaled_heatmap);
    hipDeviceSynchronize();
  }


  __global__ void heatFades(int* heatmap) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    for (int row = 0; row < SIZE; row++) {
      heatmap[row * SIZE + tid] = (int)round(heatmap[row * SIZE + tid] * 0.80);
    }
  }

  __global__ void coloringTheMap(int* heatmap, const int agents, int* desired_xs, int* desired_ys) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid > agents) return;
    if(desired_xs[tid]<0||desired_xs[tid]>=SIZE||desired_ys[tid]<0||desired_xs[tid]>=SIZE) return;

    atomicAdd(heatmap + desired_ys[tid] * SIZE + desired_xs[tid], 40);
	  atomicMin(heatmap + desired_ys[tid] * SIZE + desired_xs[tid], 255);
  }

  void Model::updateHeatmapCuda() {
    float time1, time2, time3;
    hipEvent_t fade_start, fade_stop;
    hipEventCreate(&fade_start);
    hipEventCreate(&fade_stop);
    hipEventRecord(fade_start, 0);
  
    heatFades<<<1, SIZE>>>(*heatmap);
    
    hipEventRecord(fade_stop, 0);
    hipEventSynchronize(fade_stop);
    hipEventElapsedTime(&time1, fade_start, fade_stop);
    hipEventDestroy(fade_start);
    hipEventDestroy(fade_stop);

    hipEvent_t coloring_start, coloring_stop;
    hipEventCreate(&coloring_start);
    hipEventCreate(&coloring_stop);
    hipEventRecord(coloring_start, 0);
  
    coloringTheMap<<<1, SIZE>>>(*heatmap, agents.size(), desired_xs, desired_ys);
    
    hipEventRecord(coloring_stop, 0);
    hipEventSynchronize(coloring_stop);
    hipEventElapsedTime(&time2, coloring_start, coloring_stop);
    hipEventDestroy(coloring_start);
    hipEventDestroy(coloring_stop);

    //need another gaussian filtering cuda kernel
  }

}  // namespace Ped
