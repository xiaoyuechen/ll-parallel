#include "ped_agent_soa.h"

#include "hip/hip_runtime.h"
#include ""


namespace Ped {

void* MallocPinned(std::size_t bytes) {
  void* result;
  hipHostMalloc(&result, bytes);
  return result;
}

void FreePinned(void* mem) {
  hipHostFree(mem);
}

void* MallocUnified(std::size_t bytes) {
  void* result;
  hipMallocManaged(&result, bytes);
  return result;
}

void FreeUnified(void* mem) {
  // hipFree(mem);
}

}  // namespace Ped