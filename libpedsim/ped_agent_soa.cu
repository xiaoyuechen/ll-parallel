#include "ped_agent_soa.h"

#include "hip/hip_runtime.h"
#include ""


namespace Ped {

void* MallocPinned(std::size_t bytes) {
  void* result;
  hipHostMalloc(&result, bytes);
  return result;
}

void FreePinned(void* mem) {
  hipHostFree(mem);
}

}  // namespace Ped